#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 100

#define CUDA_CHECK_RETURN(value)                                               \
  {                                                                            \
    hipError_t _m_cudaStat = value;                                           \
    if (_m_cudaStat != hipSuccess) {                                          \
      fprintf(stderr, "Error %s at line %d in file %s\n",                      \
              hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);            \
      exit(1);                                                                 \
    }                                                                          \
  }

__global__ void add_matrix_kernal(float *d_a, float *d_b, float *d_c) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int column = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < N && column < N)
    d_c[row * N + column] = d_a[row * N + column] + d_b[row * N + column];
}

void add_matrix(float *h_a, float *h_b, float *h_c) {
  float *d_a;
  float *d_b;
  float *d_c;

  CUDA_CHECK_RETURN(hipMalloc(&d_a, N * N * sizeof(float)));
  CUDA_CHECK_RETURN(hipMalloc(&d_b, N * N * sizeof(float)));
  CUDA_CHECK_RETURN(hipMalloc(&d_c, N * N * sizeof(float)));

  hipMemcpy(d_a, h_a, N * N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, N * N * sizeof(float), hipMemcpyHostToDevice);

  dim3 threads_per_block(16, 16);
  dim3 blocks_per_grid((N + threads_per_block.x - 1) / threads_per_block.x,
                       (N + threads_per_block.y - 1) / threads_per_block.y);

  add_matrix_kernal<<<blocks_per_grid, threads_per_block>>>(d_a, d_b, d_c);

  hipMemcpy(h_c, d_c, N * N * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}
int main() {
  float *h_a = (float *)malloc((N * N) * sizeof(float));
  float *h_b = (float *)malloc((N * N) * sizeof(float));
  float *h_c = (float *)malloc((N * N) * sizeof(float));

  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
      h_a[i * N + j] = (float)rand() / (RAND_MAX / 100.0);
      h_b[i * N + j] = (float)rand() / (RAND_MAX / 100.0);
    }
  }

  add_matrix(h_a, h_b, h_c);

  printf("%f + %f = %f", h_a[0], h_b[0], h_c[0]);

  return 0;
}
