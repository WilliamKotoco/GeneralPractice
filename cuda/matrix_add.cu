#include <__clang_cuda_builtin_vars.h>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 1000

// Error checking macro
#define CUDA_CHECK_RETURN(value)                                                                                       \
    {                                                                                                                  \
        hipError_t _m_cudaStat = value;                                                                               \
        if (_m_cudaStat != hipSuccess)                                                                                \
        {                                                                                                              \
            fprintf(stderr, "Error %s at line %d in file %s\n", hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);  \
            exit(1);                                                                                                   \
        }                                                                                                              \
    }

/// each thread produces one output matrix element
__device__ void add_matrix_kernal_1(float **A, float **B, float **C)
{
    int row = threadIdx.x + blockDim.x * blockIdx.x;

}

/// each thread produces one output matrix row
__device__ void add_matrix_kernal_3(float **A, float **B, float **C)
{
    int row = threadIdx.x + blockDim.x * blockIdx.x;
    
    int size = blockDim.x > N ? N : blockDim.x;

    for (int i = 0 ; i < size; i++)
    {
      C[row][i] = A[row][i] + B[row][i];
    }
}

/// each thread produces one output matrix column
__device__ void add_matrix_kernal_2(float **A, float **B, float **C)
{
    return;
}

/// not necessary the __host__ once it is by default
__host__ float **matrix_alloc()
{
    float **matrix = (float **)malloc(N * sizeof(float *));

    for (int i = 0; i < N; i++)
    {
        matrix[i] = (float *)malloc(N * sizeof(float));
    }

    return matrix;
}

float **matrix_cuda_malloc()
{
    float **matrix;
    CUDA_CHECK_RETURN(hipMalloc((void **)&matrix, N * sizeof(float *)));

    for (int i = 0; i < N; i++)
    {
        CUDA_CHECK_RETURN(hipMalloc((void **)&matrix[i], N * sizeof(float)));
    }

    return matrix;
}

void matrix_add(float **h_A, float **h_B, float **h_C)
{
    float **d_A = matrix_cuda_malloc();
    float **d_B = matrix_cuda_malloc();
    float **d_C = matrix_cuda_malloc();

    hipMemcpy(d_A, h_A, N * sizeof(float *), hipMemcpyKind::hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * sizeof(float *), hipMemcpyKind::hipMemcpyHostToDevice);



    hipMemcpy(h_C, d_C, N * sizeof(float *), hipMemcpyKind::hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
int main()
{
    float **h_A = matrix_alloc();
    float **h_B = matrix_alloc();
    float **h_C = matrix_alloc();

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            h_A[i][j] = (float)rand() / 1000;
            h_B[i][j] = (float)rand() / 1000;
        }
    }

    matrix_add(h_A, h_B, h_C);
    return 0;
}
