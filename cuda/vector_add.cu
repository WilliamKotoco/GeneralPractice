#include <cstdlib>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 10000
// Error checking macro
#define CUDA_CHECK_RETURN(value)                                                                                       \
    {                                                                                                                  \
        hipError_t _m_cudaStat = value;                                                                               \
        if (_m_cudaStat != hipSuccess)                                                                                \
        {                                                                                                              \
            fprintf(stderr, "Error %s at line %d in file %s\n", hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);  \
            exit(1);                                                                                                   \
        }                                                                                                              \
    }

/// there is no need for loops, each thread corresponds to an interation
__global__ void vec_add_kernel(float *A, float *B, float *C)
{

    /// size of block * the id identifies unically the index of the original
    /// vector
    int index = threadIdx.x + (blockDim.x * blockIdx.x);

    /// the if statemant is necessary since the size of the vector may not match
    /// the block size, once the block size is multiple of 32. It is necessary to
    /// discard the remaining threads
    if (index < N)
        C[index] = A[index] + B[index];
}
void vec_add(float *h_a, float *h_b, float *h_c)
{

    /// declaring the device arrays
    float *d_a;
    float *d_b;
    float *d_c;

    /// allocating memory in device
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_a, N * sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_b, N * sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_c, N * sizeof(float)));

    /// copying from host to device
    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);

    /// number of blocks and number of threads within a block
    vec_add_kernel<<< ceil(N/ 256.0), 256>>>(d_a, d_b, d_c);

    /// copying from device to host after performing the vector addition i
    hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);


    printf("teste %f \n", h_c[0]);
    /// free device's memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
int main()
{
    float *h_a = (float *)malloc(N * sizeof(float));
    float *h_b = (float *)malloc(N * sizeof(float));
    float *h_c = (float *)malloc(N * sizeof(float));

    for (int i = 0; i < N; i++)
    {
        h_a[i] = (float)rand() / (RAND_MAX/100.0);
        h_b[i] = (float)rand() / (RAND_MAX / 100.0);
    }

    vec_add(h_a, h_b, h_c);

    printf("Test: %f + %f = %f", h_a[0], h_b[0], h_c[0]);

    return 0;
}
