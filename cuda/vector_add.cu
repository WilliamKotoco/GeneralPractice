#include <cstdlib>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 10000

/// there is no need for loops, each thread corresponds to an interation
__global__ void vec_add_kernel(float *A, float *B, float *C)
{

    /// size of block * the id identifies unically the index of the original
    /// vector
    int index = threadIdx.x + (blockDim.x * blockIdx.x);

    /// the if statemant is necessary since the size of the vector may not match
    /// the block size, once the block size is multiple of 32. It is necessary to
    /// discard the remaining threads
    if (index < N)
        C[index] = A[index] + B[index];
}
void vec_add(float *h_a, float *h_b, float *h_c)
{

    /// declaring the device arrays
    float *d_a;
    float *d_b;
    float *d_c;

    /// allocating memory in device
    hipMalloc((void **)&d_a, N * sizeof(float));
    hipMalloc((void **)&d_b, N * sizeof(float));
    hipMalloc((void **)&d_c, N * sizeof(float));

    /// copying from host to device
    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);

    /// number of blocks and number of threads within a block
    vec_add_kernel<<<ceil(N / 256.0), 256>>>(d_a, d_b, d_c);

    /// copying from device to host after performing the vector addition i
    hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

    /// free device's memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
int main()
{
    float *h_a = (float *)malloc(N * sizeof(float));
    float *h_b = (float *)malloc(N * sizeof(float));
    float *h_c = (float *)malloc(N * sizeof(float));

    for (int i = 0; i < N; i++)
    {
        h_a[i] = (float)rand() / 10000;
        h_b[i] = (float)rand() / 10000;
    }

    vec_add(h_a, h_b, h_c);

    printf("Test: %f + %f = %f", h_a[0], h_b[0], h_c[0]);

    return 0;
}
